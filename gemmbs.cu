#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>

using namespace std;

static const char* cublasGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";

        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";

        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";

        default:
            return "<unknown>";
    }
}

inline void cublasLtCheck(hipblasStatus_t status, int iLine) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CublasLt error " << cublasGetErrorEnum(status) << " at line " << iLine << endl;
    }
}

#define cublasLtCk(call) cublasLtCheck(call, __LINE__)

float MatrixMultiply(int m, int k, int n);

void ComputeDim(string fileName)
{
    int m, k, n;
    ifstream inFile(fileName, ios::in);
    string lineStr;
    vector<vector<int> > intArray;
    getline(inFile, lineStr);
    while (getline(inFile, lineStr))
    {
        cout << lineStr << endl;
        stringstream ss(lineStr);
        string str;
        vector<int> lineArray;
        
        while (getline(ss, str, ','))
        {
            lineArray.push_back(stoi(str));
        }
        intArray.push_back(lineArray);
    }

    for (int i = 0; i < intArray.size(); ++i) {
        vector<int> line = intArray[i];
        for (int bs=1; bs <= 32; bs*=2) {
            m = bs * line[4] * line[5];
            k = line[1] * line[2] * line[3];
            n = line[0];
            cout << "convID:" << i << " bs=" << bs << " m=" << m << " k=" << k << " n=" << n << " time=" << MatrixMultiply(m, k, n) << endl;
        }
    }
    return;
}


void ConstantInit(float* data, int size, float val)
{
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

float MatrixMultiply(int m, int k, int n)
{
    float computingTime = -1;
    // Create cublas handle
    hipblasHandle_t handle;
    cublasLtCk(hipblasCreate(&handle));
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    
    // Allocate host memory for matrices A and B
    unsigned int size_A = m * k;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));

    unsigned int size_B = k * n;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));

    // Initialize host memory
    const float valB = 0.1f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate host matrix C
    unsigned int mem_size_C = m * n * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A);
    hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B);
    hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C);

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream);
    
    // Setup parameters
    // C = a(AB) + bC
    float a = 1.0f;
    float b = 0.0f;

    // Performs warmup operation
    cublasLtCk(hipblasGemmEx(handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            m,
            n,
            k,
            &a,
            d_A,
            HIP_R_32F,
            m,
            d_B,
            HIP_R_32F,
            k,
            &b,
            d_C,
            HIP_R_32F,
            m,
            HIPBLAS_COMPUTE_32F,
            CUBLAS_GEMM_AlGO0
        ));
    hipStreamSynchronize(stream);
    // Record the start event
    hipEventRecord(start, stream);
    int nIter = 20;
    for (int i = 0; i < nIter; ++i) {
        cublasLtCk(hipblasGemmEx(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                m,
                n,
                k,
                &a,
                d_A,
                HIP_R_32F,
                m,
                d_B,
                HIP_R_32F,
                k,
                &b,
                d_C,
                HIP_R_32F,
                m,
                HIPBLAS_COMPUTE_32F,
                CUBLAS_GEMM_ALGO0
            ));
    }
    // Record the stop event
    hipEventRecord(stop, stream);
    
    // Wait for the stop event to complete
    hipEventSynchronize(stop);
    hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    float msecPerMatrixMul = msecTotal / nIter;
    computingTime = msecPerMatrixMul * 1000;

    // Print the result matrix
      
    // std::cout << "Time: " << msec << " ms" << std::endl; 

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return computingTime; 
    
}

int main()
{
    ComputeDim("resnet50_conv.csv");
    // cout << MatrixMultiply(64, 100, 100) << endl;
    // cout << MatrixMultiply(128, 100, 100) << endl;
}

